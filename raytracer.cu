#include "hip/hip_runtime.h"
#include "raytracer.h"
#include "hip/hip_vector_types.h"

#define BLOCK_SIZE 16     // block size

static uchar4* kernelOutputBuffer;
static int g_screenWidth;
static int g_screenHeight;
static size_t g_bufferPitch;

__device__ float IntBound(float s, float ds) {
    return (ds > 0 ? ceil(s) - s : s - floor(s)) / abs(ds);
}

// Returns true if there was an intersection.
__device__ bool TraverseRenderChunk(void** devicePointers, float3 origin, float3 direction, float* distance) {


    return false;
}

__device__ bool IntersectQuad(float3 ray, Quad quad, float* out_distance) {
    return false;
}

__global__ void Kernel(uchar4* dst, int width, int height, void** devicePointers, int* arraySizes, Viewport viewport, float3 entity, size_t bufferPitch) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    // Invert Y because OpenGL
    int y = height - ((blockIdx.y * blockDim.y) + threadIdx.y + 1);

    float u = x / (float)width;
    float v = y / (float)height;

    int offset = (y * bufferPitch) + x * sizeof(uchar4);
    if (offset >= bufferPitch * height) return;

    float3 point = (lerp(viewport.p0, viewport.p1, u) + lerp(viewport.p0, viewport.p2, v)) * 0.5f;
    float3 direction = normalize(point - viewport.origin);
    float3 origin = entity + viewport.origin;

    float distance = FLT_MAX;

    int renderChunkX = MAX_RENDER_DISTANCE;
    int renderChunkY = floor(origin.y) / 16;
    int renderChunkZ = MAX_RENDER_DISTANCE;

    // = (-1/1) signs of vector dir
    int stepX = (direction.x < 0) ? -1 : 1;
    int stepY = (direction.y < 0) ? -1 : 1;
    int stepZ = (direction.z < 0) ? -1 : 1;

    float tMaxX = IntBound(origin.x, direction.x);
    float tMaxY = IntBound(origin.y, direction.y);
    float tMaxZ = IntBound(origin.z, direction.z);

    // TODO: Save registers
    float tDeltaX = (float)stepX / direction.x; // length of v between two YZ-boundaries
    float tDeltaY = (float)stepY / direction.y; // length of v between two XZ-boundaries
    float tDeltaZ = (float)stepZ / direction.z; // length of v between two XY-boundaries

    // Range of 5 renderChunks for now
    do {
        if (tMaxX < tMaxY) {
            if (tMaxX < tMaxZ) {
                renderChunkX += stepX;
                if (renderChunkX < 0 || renderChunkX >= GRID_DIM) break;
                tMaxX += tDeltaX;
            }
            else {
                renderChunkZ += stepZ;
                if (renderChunkZ < 0 || renderChunkZ >= GRID_DIM) break;
                tMaxZ += tDeltaZ;
            }
        }
        else {
            if (tMaxY < tMaxZ) {
                renderChunkY += stepY;
                if (renderChunkY < 0 || renderChunkY >= 16) break;
                tMaxY += tDeltaY;
            }
            else {
                renderChunkZ += stepZ;
                if (renderChunkZ < 0 || renderChunkZ >= GRID_DIM) break;
                tMaxZ += tDeltaZ;
            }
        }

        int devPtrOffset =
            renderChunkX * GRID_DIM * 16 * 4 +
            renderChunkZ * 16 * 4 +
            renderChunkY * 4;

        // Create the ray used for intersection
        float3 ray{
            (tMaxX - (int)tMaxX) * 16,
            (tMaxY - (int)tMaxY) * 16,
            (tMaxZ - (int)tMaxZ) * 16,
        };

        if (devPtrOffset >= DEVICE_PTRS_COUNT) break;

        void** ptr = devicePointers + devPtrOffset;

        for (int i = 0; i < 4; i++) {
            // Buffers in RenderChunk
            Quad* buffer = (Quad*)ptr[i];
            if (buffer) {
                for (int j = 0; j < arraySizes[devPtrOffset + i]; j++) {
                    // Quads in buffer
                    float dist;
                    if (IntersectQuad(ray, buffer[j], &dist)) {
                        if (dist < distance) {
                            // TODO: Remember quad for texturing etc
                            distance = dist;
                        }
                    }
                }
            }
        }

        if (distance != FLT_MAX) break;
    } while (true);

    unsigned char val = distance != FLT_MAX ? 255 : 0;

    //*((uchar4*)(((uchar1*)dst) + offset)) = make_uchar4(u * 256.0f, v * 256.0f, 255.0f, 255.0f);
    *((uchar4*)(((uchar1*)dst) + offset)) = make_uchar4(val, val, 255, 255);
}

void rtResize(JNIEnv* env, int screenWidth, int screenHeight) {
    g_screenWidth = screenWidth;
    g_screenHeight = screenHeight;

    hipError_t err;
    
    // Resize
    if (kernelOutputBuffer) {
        err = hipFree(kernelOutputBuffer);
        if (err != hipSuccess) {
            Log(env, std::string("hipFree failed: ") + std::to_string(err));
        }
    }

    err = hipMallocPitch((void**)&kernelOutputBuffer, &g_bufferPitch, g_screenWidth * sizeof(uchar4), g_screenHeight * sizeof(uchar4));
    if (err != hipSuccess) {
        Log(env, std::string("hipMalloc failed: ") + std::to_string(err));
    }
}

void rtRaytrace(JNIEnv*, hipGraphicsResource_t glTexture, int texHeight, void** devicePointers, int* arraySizes, const Viewport &viewport, const float3& viewEntity) {
    unsigned int blocksW = (unsigned int)ceilf(g_screenWidth / (float)BLOCK_SIZE);
    unsigned int blocksH = (unsigned int)ceilf(g_screenHeight / (float)BLOCK_SIZE);
    dim3 gridDim(blocksW, blocksH, 1);
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);

    // Kernel call
    Kernel<<<gridDim, blockDim>>>(kernelOutputBuffer, g_screenWidth, g_screenHeight, devicePointers, arraySizes, viewport, viewEntity, g_bufferPitch);

    // Copy CUDA result to OpenGL texture
    hipArray* mappedGLArray;
    hipGraphicsSubResourceGetMappedArray(&mappedGLArray, glTexture, 0, 0);

    int width = g_screenWidth * sizeof(uchar4);
    hipMemcpy2DToArray(
        mappedGLArray,              // dst
        0,                          // wOffset
        texHeight - g_screenHeight, // hOffset
        kernelOutputBuffer,         // src
        g_bufferPitch,              // spitch
        width,                      // width
        g_screenHeight,             // height
        hipMemcpyDeviceToDevice    // kind
    );
}
