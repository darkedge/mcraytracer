#include "hip/hip_runtime.h"
#include "raytracer.h"
#include "hip/hip_vector_types.h"
#include <>

// BLOCK_SIZE^2 = max threads per SM / max active blocks
#define BLOCK_SIZE 8 // JOTARO
#define EPSILON 0.000001f

#define USE_INTRINSICS 0

static uchar4* kernelOutputBuffer;
static int g_screenWidth;
static int g_screenHeight;
static size_t g_bufferPitch;

// Calculates t for a line starting from s
// to cross the next integer in terms of ds.
// Assume s = [0..1], ds is [-1..1]
inline __device__ float FindFirstT(float s, float ds) {
    return (ds > 0 ? ceilf(s) - s : s - floorf(s)) / fabsf(ds);
}

// Transforms a point from world space to grid space [0..1].
static float3 WorldToGrid(float3 f) {
    return fracf(f * (1 / 16.0f));
}

// https://tavianator.com/fast-branchless-raybounding-box-intersections/
__device__ bool IntersectRayAABB(float3 origin, float3 dirInv, char4 chunk, char i, float extents) {
    float3 min, max;
    min.x = (float)chunk.x + (i & 1) * extents;
    min.y = (float)chunk.y + ((i >> 1) & 1) * extents;
    min.z = (float)chunk.z + (i >> 2) * extents;

    max.x = min.x + extents;
    max.y = min.y + extents;
    max.z = min.z + extents;

    float t1 = (min.x - origin.x)*dirInv.x;
    float t2 = (max.x - origin.x)*dirInv.x;

    float tmin = fminf(t1, t2);
    float tmax = fmaxf(t1, t2);

    t1 = (min.y - origin.y)*dirInv.y;
    t2 = (max.y - origin.y)*dirInv.y;

    tmin = fmaxf(tmin, fminf(t1, t2));
    tmax = fminf(tmax, fmaxf(t1, t2));

    t1 = (min.x - origin.z)*dirInv.z;
    t2 = (max.x - origin.z)*dirInv.z;

    tmin = fmaxf(tmin, fminf(t1, t2));
    tmax = fminf(tmax, fmaxf(t1, t2));

    return tmax >= tmin;
}

__global__ void Kernel(uchar4* dst, int width, int height, size_t bufferPitch, const DevicePointers* __restrict__ vertexBuffers, Viewport viewport, float3 origin, char renderChunkY) {
    float3 direction;
    {
        int x = (blockIdx.x * blockDim.x) + threadIdx.x;
        // Invert Y because OpenGL
        int y = height - ((blockIdx.y * blockDim.y) + threadIdx.y + 1);
        float u = x / (float)width;
        float v = y / (float)height;

        float3 point = lerp(viewport.p0, viewport.p1, u) + lerp(viewport.p0, viewport.p2, v) - viewport.p0;
        direction = normalize(point - viewport.origin);
    }

    // = (-1/1) signs of vector dir
    char4 step = make_char4(
        (direction.x < 0) ? -1 : 1,
        (direction.y < 0) ? -1 : 1,
        (direction.z < 0) ? -1 : 1,
        0
    );

    float3 dirInv = 1.0f / direction;

    // All positive values
    float tMaxX = FindFirstT(origin.x, direction.x);
    float tMaxY = FindFirstT(origin.y, direction.y);
    float tMaxZ = FindFirstT(origin.z, direction.z);

    float3 raypos = origin * 16.0f;

    char4 renderChunk = make_char4(
        MAX_RENDER_DISTANCE,
        renderChunkY,
        MAX_RENDER_DISTANCE,
        threadIdx.y * BLOCK_SIZE + threadIdx.x
    );

    float distance = FLT_MAX;
    while (true) {
        int index =
            (renderChunk.x * GRID_DIM << 4) +
            (renderChunk.z << 4) +
            renderChunk.y;

        // Get octree at this RenderChunk
        int* octree = (int*) vertexBuffers[index].octree;
        if (octree) {
            // Assume this exists
            Quad* quads = (Quad*) vertexBuffers[index].vertexBuffer;
            int* head = octree;
            int offset;

            // Traverse octree
            char4 abcd = make_char4(0, 0, 0, 0);
            for (; abcd.x < 8; abcd.x++) {
                offset = head[abcd.x];
                if (offset != 0 && IntersectRayAABB(raypos, dirInv, renderChunk, abcd.x, 8.0f)) {
                    head = octree + offset;
                    for (; abcd.y < 8; abcd.y++) {
                        offset = head[abcd.y];
                        if (offset != 0 && IntersectRayAABB(raypos, dirInv, renderChunk, abcd.y, 4.0f)) {
                            head = octree + offset;
                            for (; abcd.z < 8; abcd.z++) {
                                offset = head[abcd.z];
                                if (offset != 0 && IntersectRayAABB(raypos, dirInv, renderChunk, abcd.z, 2.0f)) {
                                    head = octree + offset;
                                    for (; abcd.w < 8; abcd.w++) {
                                        offset = head[abcd.w];
                                        if (offset != 0 && IntersectRayAABB(raypos, dirInv, renderChunk, abcd.w, 1.0f)) {
                                            head = octree + offset;
                                            for (int i = 1; i < head[0]; i++) {
                                                Quad *q = &quads[head[i]];
                                                // Triangle 1
                                                float3 v0v1 = q->v1.pos - q->v0.pos; // e1
                                                float3 v0v2 = q->v2.pos - q->v0.pos; // e2
                                                float3 pvec = cross(direction, v0v2); // P
                                                float det = dot(v0v1, pvec);

                                                if (det < EPSILON) continue; // Ray does not hit front face

                                                det = 1.0f / det;

                                                float3 tvec = raypos - q->v0.pos;
                                                float u = dot(tvec, pvec) * det;
                                                if (!(u < 0.0f || u > 1.0f)) {
                                                    float3 qvec = cross(tvec, v0v1);
                                                    float v = dot(direction, qvec) * det;

                                                    if (!(v < 0.0f || u + v > 1.0f)) {
                                                        float dist = dot(v0v2, qvec) * det;

                                                        if (dist < distance) {
                                                            distance = dist;
                                                        }

                                                        // Found a hit
                                                        continue;
                                                    }
                                                }

                                                // Triangle 2
                                                // TODO: Optimize this further
                                                det = -det;
                                                tvec = raypos - q->v2.pos;
                                                u = dot(tvec, pvec) * det;

                                                if (!(u < 0.0f || u > 1.0f)) {

                                                    float3 qvec = cross(tvec, v0v1);
                                                    float v = dot(direction, qvec) * det;

                                                    if (!(v < 0.0f || u + v > 1.0f)) {

                                                        float dist = dot(v0v2, qvec) * det;

                                                        if (dist < distance) {
                                                            distance = dist;
                                                        }
                                                    }
                                                }
                                            }
                                            if (distance != FLT_MAX) goto done;
                                        }
                                    }
                                }
                            }
                        }
                    }
                }
            }
        }

#define TODO_RENDER_DISTANCE 1
        if (tMaxX < tMaxY) {
            if (tMaxX < tMaxZ) {
                renderChunk.x += step.x;
                if (renderChunk.x < (MAX_RENDER_DISTANCE - TODO_RENDER_DISTANCE) || (renderChunk.x > MAX_RENDER_DISTANCE + TODO_RENDER_DISTANCE)) break;
                tMaxX += step.x / direction.x;
#if USE_INTRINSICS
                raypos.x = __fmaf_rn(tMaxX, direction.x, origin.x);
                raypos.y = __fmaf_rn(tMaxX, direction.y, origin.y);
                raypos.z = __fmaf_rn(tMaxX, direction.z, origin.z);
#else
                raypos = tMaxX * direction + origin;
#endif
            }
            else {
                renderChunk.z += step.z;
                if (renderChunk.z < (MAX_RENDER_DISTANCE - TODO_RENDER_DISTANCE) || (renderChunk.z > MAX_RENDER_DISTANCE + TODO_RENDER_DISTANCE)) break;
                tMaxZ += step.z / direction.z;
#if USE_INTRINSICS
                raypos.x = __fmaf_rn(tMaxZ, direction.x, origin.x);
                raypos.y = __fmaf_rn(tMaxZ, direction.y, origin.y);
                raypos.z = __fmaf_rn(tMaxZ, direction.z, origin.z);
#else
                raypos = tMaxZ * direction + origin;
#endif
            }
        } else {
            if (tMaxY < tMaxZ) {
                renderChunk.y += step.y;
                if (renderChunk.y < 0 || renderChunk.y >= 16) break;
                tMaxY += step.y / direction.y;
#if USE_INTRINSICS
                raypos.x = __fmaf_rn(tMaxY, direction.x, origin.x);
                raypos.y = __fmaf_rn(tMaxY, direction.y, origin.y);
                raypos.z = __fmaf_rn(tMaxY, direction.z, origin.z);
#else
                raypos = tMaxY * direction + origin;
#endif
            }
            else {
                renderChunk.z += step.z;
                if (renderChunk.z < (MAX_RENDER_DISTANCE - TODO_RENDER_DISTANCE) || (renderChunk.z > MAX_RENDER_DISTANCE + TODO_RENDER_DISTANCE)) break;
                tMaxZ += step.z / direction.z;
#if USE_INTRINSICS
                raypos.x = __fmaf_rn(tMaxZ, direction.x, origin.x);
                raypos.y = __fmaf_rn(tMaxZ, direction.y, origin.y);
                raypos.z = __fmaf_rn(tMaxZ, direction.z, origin.z);
#else
                raypos = tMaxZ * direction + origin;
#endif
            }
        }

        raypos = fracf(raypos) * 16.0f;
    }

    done:

    {
        int x = (blockIdx.x * blockDim.x) + threadIdx.x;
        // Invert Y because OpenGL
        int y = height - ((blockIdx.y * blockDim.y) + threadIdx.y + 1);
        int offset = (y * bufferPitch) + (x << 2); // x * sizeof(uchar4)
        if (offset >= bufferPitch * height) return;
        dst = (uchar4*)(((char*)dst) + offset);
    }

    unsigned char val = distance != FLT_MAX ? 255 : 0;
    *dst = make_uchar4(val, 0, 255, 255);
}

void rtResize(JNIEnv* env, int screenWidth, int screenHeight) {
    g_screenWidth = screenWidth;
    g_screenHeight = screenHeight;

    // Resize
    if (kernelOutputBuffer) {
        CUDA_TRY(hipFree(kernelOutputBuffer));
    }

    CUDA_TRY(hipMallocPitch((void**)&kernelOutputBuffer, &g_bufferPitch, g_screenWidth * sizeof(uchar4), g_screenHeight * sizeof(uchar4)));
}

void rtRaytrace(JNIEnv*, hipGraphicsResource_t glTexture, int texHeight, void* devicePointers, const Viewport &viewport, const float3& viewEntity) {
    unsigned int blocksW = (unsigned int)ceilf(g_screenWidth / (float)BLOCK_SIZE);
    unsigned int blocksH = (unsigned int)ceilf(g_screenHeight / (float)BLOCK_SIZE);
    dim3 gridDim(blocksW, blocksH, 1);
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);

    // Kernel call
#if 0
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    Kernel<<<gridDim, blockDim>>>(kernelOutputBuffer, g_screenWidth, g_screenHeight, devicePointers, viewport, viewEntity, g_bufferPitch);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    Log(env, std::to_string(time));
#endif

    // World space
    float3 origin = viewEntity + viewport.origin;
    char renderChunkY = (char)floorf(origin.y / 16.0f);

    // Transform origin to [0..1]
    origin = WorldToGrid(origin);
    Kernel<<<gridDim, blockDim>>>(kernelOutputBuffer, g_screenWidth, g_screenHeight, g_bufferPitch, (const DevicePointers*)devicePointers, viewport, origin, renderChunkY);

    // Copy CUDA result to OpenGL texture
    hipArray* mappedGLArray;
    hipGraphicsSubResourceGetMappedArray(&mappedGLArray, glTexture, 0, 0);

    int width = g_screenWidth * sizeof(uchar4);
    hipMemcpy2DToArray(
        mappedGLArray,              // dst
        0,                          // wOffset
        texHeight - g_screenHeight, // hOffset
        kernelOutputBuffer,         // src
        g_bufferPitch,              // spitch
        width,                      // width
        g_screenHeight,             // height
        hipMemcpyDeviceToDevice    // kind
    );
}
